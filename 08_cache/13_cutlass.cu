
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  const int ItemsPerVector = 8;
  const int ItemsPerThread = ItemsPerVector; // 8

  const int ThreadsPerWarpY = 4;
  const int ThreadsPerWarpX = 8;
  const int ThreadsPerWarp = ThreadsPerWarpX * ThreadsPerWarpY; // 32

  const int ItemsPerWarpY = ThreadsPerWarpY * ItemsPerThread; // 32
  const int ItemsPerWarpX = ThreadsPerWarpX * ItemsPerThread; // 64

  const int Ktile = 8;

  int offset_a_m = ItemsPerWarpX * blockIdx.x / ItemsPerVector;
  int offset_b_n = ItemsPerWarpX * blockIdx.y;
  int lda = dim_m / ItemsPerVector;
  int ldb = dim_k / ItemsPerVector;
  int a_m = threadIdx.x % ThreadsPerWarpX; // 16
  int a_k = threadIdx.x / ThreadsPerWarpX; // 4

  struct __align__(16) vec_t { float d[ItemsPerVector]; };
  vec_t *tile_a;
  vec_t *tile_b;
  vec_t __align__(16) thread_a;
  vec_t __align__(16) thread_b;
  __shared__ float __align__(16) block_a[Ktile][ItemsPerWarpX];
  __shared__ float __align__(16) block_b[Ktile][ItemsPerWarpX];
  float __align__(16) fragment_a[ItemsPerThread];
  float __align__(16) fragment_b[ItemsPerThread];
  float __align__(16) fragment_c[ItemsPerThread][ItemsPerThread];

  tile_a = reinterpret_cast<vec_t*>(&d_a[(a_k * lda + (a_m + offset_a_m)) * ItemsPerVector]);
  tile_b = reinterpret_cast<vec_t*>(&d_b[((threadIdx.x + offset_b_n) * ldb) * ItemsPerVector]);
  for (int m = 0; m < ItemsPerThread; ++m)
    for (int n = 0; n < ItemsPerThread; ++n)
      fragment_c[m][n] = 0;

  int warp_id = threadIdx.x / ThreadsPerWarp; // 2
  int lane_id = threadIdx.x % ThreadsPerWarp; // 32
  int lane_x = lane_id / ThreadsPerWarpY; // 8
  int lane_y = lane_id % ThreadsPerWarpY; // 4
  int offset_y = lane_y * ItemsPerVector + warp_id * ItemsPerWarpY;
  int offset_x = lane_x * ItemsPerVector;
  int stride_a = 0;
  int stride_b = 0;
  for (int kk = 0; kk < dim_k; kk += Ktile) {
    thread_a = tile_a[stride_a];
    thread_b = tile_b[stride_b];
    __syncthreads();
    for (int j = 0; j < ItemsPerVector; ++j) {
      block_a[a_k][a_m * ItemsPerVector + j] = thread_a.d[j];
      block_b[j][threadIdx.x] = thread_b.d[j];
    }
    __syncthreads();
    stride_a += lda * Ktile;
    stride_b += Ktile / ItemsPerVector;
#pragma unroll
    for (int k = 0; k < Ktile; k++) {
      for (int j = 0; j < ItemsPerVector; ++j) {
	fragment_a[j] = block_a[k][offset_y + j];
	fragment_b[j] = block_b[k][offset_x + j];
      }
      for (int m = 0; m < ItemsPerThread; ++m) {
	for (int n = 0; n < ItemsPerThread; ++n) {
	  fragment_c[m][n] += fragment_a[m] * fragment_b[n];
	}
      }
    }
  }
  for (int ix = 0; ix < ItemsPerThread; ++ix) {
    for (int iy = 0; iy < ItemsPerThread; iy += ItemsPerVector) {
      int vx = ix / ItemsPerVector;
      int vy = iy / ItemsPerVector;
      int tx = offset_x + (vx * ThreadsPerWarpX * ItemsPerVector) + (ix % ItemsPerVector);
      int ty = offset_y + (vy * ThreadsPerWarpY * ItemsPerVector) + (iy % ItemsPerVector);
      int bx = ItemsPerWarpX * blockIdx.y + tx;
      int by = ItemsPerWarpX * blockIdx.x + ty;
      for (int i = 0; i < ItemsPerVector; ++i) {
	if (bx < dim_n && (by + i) < dim_m) {
	  d_c[bx * dim_m + by + i] = fragment_c[iy + i][ix];
	}
      }
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}
